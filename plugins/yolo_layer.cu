#include "hip/hip_runtime.h"
/*
 * yolo_layer.cu
 *
 * This code was originally written by wang-xinyu under MIT license.
 * I took it from:
 *
 *     https://github.com/wang-xinyu/tensorrtx/tree/master/yolov4
 *
 * and made necessary modifications.
 *
 * - JK Jung
 */

#include "yolo_layer.h"

using namespace Yolo;

namespace
{
// Write values into buffer
template <typename T>
void write(char*& buffer, const T& val)
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

// Read values from buffer
template <typename T>
void read(const char*& buffer, T& val)
{
    val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
}
} // namespace

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int yolo_width, int yolo_height, int num_anchors, float* anchors, int num_classes, int input_width, int input_height, float scale_x_y, int new_coords)
    {
        mYoloWidth   = yolo_width;
        mYoloHeight  = yolo_height;
        mNumAnchors  = num_anchors;
        memcpy(mAnchorsHost, anchors, num_anchors * 2 * sizeof(float));
        mNumClasses  = num_classes;
        mInputWidth  = input_width;
        mInputHeight = input_height;
        mScaleXY     = scale_x_y;
        mNewCoords   = new_coords;

        CHECK(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
        CHECK(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));
    }

    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mThreadCount);
        read(d, mYoloWidth);
        read(d, mYoloHeight);
        read(d, mNumAnchors);
        memcpy(mAnchorsHost, d, MAX_ANCHORS * 2 * sizeof(float));
        d += MAX_ANCHORS * 2 * sizeof(float);
        read(d, mNumClasses);
        read(d, mInputWidth);
        read(d, mInputHeight);
        read(d, mScaleXY);
        read(d, mNewCoords);

        CHECK(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
        CHECK(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));

        assert(d == a + length);
    }

    IPluginV2DynamicExt* YoloLayerPlugin::clone() const NOEXCEPT
    {
        YoloLayerPlugin *p = new YoloLayerPlugin(mYoloWidth, mYoloHeight, mNumAnchors, (float*) mAnchorsHost, mNumClasses, mInputWidth, mInputHeight, mScaleXY, mNewCoords);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    void YoloLayerPlugin::terminate() NOEXCEPT
    {
        CHECK(hipFree(mAnchors));
    }

    size_t YoloLayerPlugin::getSerializationSize() const NOEXCEPT
    {
        return sizeof(mThreadCount) + \
               sizeof(mYoloWidth) + sizeof(mYoloHeight) + \
               sizeof(mNumAnchors) + MAX_ANCHORS * 2 * sizeof(float) + \
               sizeof(mNumClasses) + \
               sizeof(mInputWidth) + sizeof(mInputHeight) + \
               sizeof(mScaleXY) + sizeof(mNewCoords);
    }

    void YoloLayerPlugin::serialize(void* buffer) const NOEXCEPT
    {
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mThreadCount);
        write(d, mYoloWidth);
        write(d, mYoloHeight);
        write(d, mNumAnchors);
        memcpy(d, mAnchorsHost, MAX_ANCHORS * 2 * sizeof(float));
        d += MAX_ANCHORS * 2 * sizeof(float);
        write(d, mNumClasses);
        write(d, mInputWidth);
        write(d, mInputHeight);
        write(d, mScaleXY);
        write(d, mNewCoords);

        assert(d == a + getSerializationSize());
    }

    DimsExprs YoloLayerPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, int32_t nbInputs, IExprBuilder &exprBuilder) NOEXCEPT
    {
        assert(outputIndex == 0);
        assert(nbInputs == 1);
        assert(inputs[0].d[1]->isConstant() && inputs[0].d[1]->getConstantValue() == (mNumClasses + 5) * mNumAnchors);
        assert(inputs[0].d[2]->isConstant() && inputs[0].d[2]->getConstantValue() == mYoloHeight);
        assert(inputs[0].d[3]->isConstant() && inputs[0].d[3]->getConstantValue() == mYoloWidth);

        DimsExprs output(inputs[0]);

        // output detection results to the channel dimension
        int totalsize = mYoloWidth * mYoloHeight * mNumAnchors * sizeof(Detection) / sizeof(float);

        output.d[1] = exprBuilder.constant(totalsize);
        output.d[2] = exprBuilder.constant(1);
        output.d[3] = exprBuilder.constant(1);
        return output;
    }

    inline __device__ float sigmoidGPU(float x) { return 1.0f / (1.0f + __expf(-x)); }

    inline __device__ float scale_sigmoidGPU(float x, float s)
    {
        return s * sigmoidGPU(x) - (s - 1.0f) * 0.5f;
    }

    // CalDetection(): This kernel processes 1 yolo layer calculation.  It
    // distributes calculations so that 1 GPU thread would be responsible
    // for each grid/anchor combination.
    // NOTE: The output (x, y, w, h) are between 0.0 and 1.0
    //       (relative to orginal image width and height).
    __global__ void CalDetection(const float *input, float *output,
                                 int batch_size,
                                 int yolo_width, int yolo_height,
                                 int num_anchors, const float *anchors,
                                 int num_classes, int input_w, int input_h,
                                 float scale_x_y)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        Detection* det = ((Detection*) output) + idx;
        int total_grids = yolo_width * yolo_height;
        if (idx >= batch_size * total_grids * num_anchors) return;

        int info_len = 5 + num_classes;
        //int batch_idx = idx / (total_grids * num_anchors);
        int group_idx = idx / total_grids;
        int anchor_idx = group_idx % num_anchors;
        const float* cur_input = input + group_idx * (info_len * total_grids) + (idx % total_grids);

        int class_id;
        float max_cls_logit = -HIP_INF_F;  // minus infinity
        for (int i = 5; i < info_len; ++i) {
            float l = *(cur_input + i * total_grids);
            if (l > max_cls_logit) {
                max_cls_logit = l;
                class_id = i - 5;
            }
        }
        float max_cls_prob = sigmoidGPU(max_cls_logit);
        float box_prob = sigmoidGPU(*(cur_input + 4 * total_grids));
        //if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH)
        //    return;

        int row = (idx % total_grids) / yolo_width;
        int col = (idx % total_grids) % yolo_width;

        det->bbox[0] = (col + scale_sigmoidGPU(*(cur_input + 0 * total_grids), scale_x_y)) / yolo_width;    // [0, 1]
        det->bbox[1] = (row + scale_sigmoidGPU(*(cur_input + 1 * total_grids), scale_x_y)) / yolo_height;   // [0, 1]
        det->bbox[2] = __expf(*(cur_input + 2 * total_grids)) * *(anchors + 2 * anchor_idx + 0) / input_w;  // [0, 1]
        det->bbox[3] = __expf(*(cur_input + 3 * total_grids)) * *(anchors + 2 * anchor_idx + 1) / input_h;  // [0, 1]

        det->bbox[0] -= det->bbox[2] / 2;  // shift from center to top-left
        det->bbox[1] -= det->bbox[3] / 2;

        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }

    inline __device__ float scale(float x, float s)
    {
        return s * x - (s - 1.0f) * 0.5f;
    }

    inline __device__ float square(float x)
    {
        return x * x;
    }

    __global__ void CalDetection_NewCoords(const float *input, float *output,
                                           int batch_size,
                                           int yolo_width, int yolo_height,
                                           int num_anchors, const float *anchors,
                                           int num_classes, int input_w, int input_h,
                                           float scale_x_y)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        Detection* det = ((Detection*) output) + idx;
        int total_grids = yolo_width * yolo_height;
        if (idx >= batch_size * total_grids * num_anchors) return;

        int info_len = 5 + num_classes;
        //int batch_idx = idx / (total_grids * num_anchors);
        int group_idx = idx / total_grids;
        int anchor_idx = group_idx % num_anchors;
        const float* cur_input = input + group_idx * (info_len * total_grids) + (idx % total_grids);

        int class_id;
        float max_cls_prob = -HIP_INF_F;  // minus infinity
        for (int i = 5; i < info_len; ++i) {
            float l = *(cur_input + i * total_grids);
            if (l > max_cls_prob) {
                max_cls_prob = l;
                class_id = i - 5;
            }
        }
        float box_prob = *(cur_input + 4 * total_grids);
        //if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH)
        //    return;

        int row = (idx % total_grids) / yolo_width;
        int col = (idx % total_grids) % yolo_width;

        det->bbox[0] = (col + scale(*(cur_input + 0 * total_grids), scale_x_y)) / yolo_width;                   // [0, 1]
        det->bbox[1] = (row + scale(*(cur_input + 1 * total_grids), scale_x_y)) / yolo_height;                  // [0, 1]
        det->bbox[2] = square(*(cur_input + 2 * total_grids)) * 4 * *(anchors + 2 * anchor_idx + 0) / input_w;  // [0, 1]
        det->bbox[3] = square(*(cur_input + 3 * total_grids)) * 4 * *(anchors + 2 * anchor_idx + 1) / input_h;  // [0, 1]

        det->bbox[0] -= det->bbox[2] / 2;  // shift from center to top-left
        det->bbox[1] -= det->bbox[3] / 2;

        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }

    void YoloLayerPlugin::forwardGpu(const float* const* inputs, float* output, hipStream_t stream, int batchSize)
    {
        int num_elements = batchSize * mNumAnchors * mYoloWidth * mYoloHeight;

        //CHECK(hipMemset(output, 0, num_elements * sizeof(Detection)));

        if (mNewCoords) {
            CalDetection_NewCoords<<<(num_elements + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[0], output, batchSize, mYoloWidth, mYoloHeight, mNumAnchors, (const float*) mAnchors, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
        } else {
            CalDetection<<<(num_elements + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[0], output, batchSize, mYoloWidth, mYoloHeight, mNumAnchors, (const float*) mAnchors, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
        }
    }

#if NV_TENSORRT_MAJOR == 8
    int32_t YoloLayerPlugin::enqueue(int32_t batchSize, void const* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) NOEXCEPT
#else
    int YoloLayerPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream)
#endif
    {
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, inputDesc->dims.d[0]);
        return 0;
    }

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const NOEXCEPT
    {
        return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const NOEXCEPT
    {
        return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames() NOEXCEPT
    {
        return &mFC;
    }

    IPluginV2DynamicExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) NOEXCEPT
    {
        assert(!strcmp(name, getPluginName()));
        const PluginField* fields = fc->fields;
        int yolo_width, yolo_height, num_anchors = 0;
        float anchors[MAX_ANCHORS * 2];
        int num_classes, input_multiplier, new_coords = 0;
        float scale_x_y = 1.0;

        for (int i = 0; i < fc->nbFields; ++i)
        {
            const char* attrName = fields[i].name;
            if (!strcmp(attrName, "yoloWidth"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                yolo_width = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "yoloHeight"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                yolo_height = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "numAnchors"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                num_anchors = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "numClasses"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                num_classes = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "inputMultiplier"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                input_multiplier = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "anchors")){
                assert(num_anchors > 0 && num_anchors <= MAX_ANCHORS);
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                memcpy(anchors, static_cast<const float*>(fields[i].data), num_anchors * 2 * sizeof(float));
            }
            else if (!strcmp(attrName, "scaleXY"))
            {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                scale_x_y = *(static_cast<const float*>(fields[i].data));
            }
            else if (!strcmp(attrName, "newCoords"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                new_coords = *(static_cast<const int*>(fields[i].data));
            }
            else
            {
                std::cerr <<  "Unknown attribute: " << attrName << std::endl;
                assert(0);
            }
        }
        assert(yolo_width > 0 && yolo_height > 0);
        assert(anchors[0] > 0.0f && anchors[1] > 0.0f);
        assert(num_classes > 0);
        assert(input_multiplier == 64 || input_multiplier == 32 || \
               input_multiplier == 16 || input_multiplier == 8);
        assert(scale_x_y >= 1.0);

        YoloLayerPlugin* obj = new YoloLayerPlugin(yolo_width, yolo_height, num_anchors, anchors, num_classes, yolo_width * input_multiplier, yolo_height * input_multiplier, scale_x_y, new_coords);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2DynamicExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) NOEXCEPT
    {
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;
} // namespace nvinfer1
